#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdint>
#include <libgpuc_cuda.hpp>
#include <string>

template <typename T>
class Mean {
public:
  T mean = 0;
  uint32_t n = 0;

  __global__ void comsume(T sample) {
    n++;
    T delta = sample - mean;
    mean += delta / n;
  }

  __global__ void merge(const Mean<T>& other) {
    if (other.n == 0) {
      return;
    }
    if (n == 0) {
      mean = other.mean;
      n = other.n;
      return;
    }

    n = n + other.n;
    T delta = other.mean - mean;
    mean += delta * other.n / n;
  }
}

template <typename T>
__global__ void mean2DKernel(T* out, T* in, uint32_t numCols) {
  uint32_t numThreads = blockDim.x;
  // uint32_t numRows = gridDim.y;
  uint32_t row = blockIdx.x;
  Mean record();
  for (uint32_t col = threadIdx.x; col < numCols; col += numThreads) {
    uint32_t idx = row * numCols + col;
    record.comsume(in[idx]);
  }
  __syncthreads();

  // Do warp reduction
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    record = record.merge(__shfl_down_sync(0xffffffff, record, offset));
  }
  __syncthreads();

  uint8_t lane = threadIdx.x % warpSize;
  uint8_t warp = threadIdx.x / warpSize;

  __shared__ Mean<T> sdata[32];

  if (lane == 0) {
    sdata[warp] = record;
  }
  __syncthreads();

  if (warp == 0) {
    sum = (lane < blockDim.x / warpSize) ? sdata[lane] : 0;
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
      record = record.merge(__shfl_down_sync(0xffffffff, record, offset));
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    out[row] = sum.mean;
  }
}

void mean2DTensor(Tensor out, Tensor in) {
  if (in.ndim != 2) {
    throw std::string("Input tensor must be 2D");
  } else if (out.ndim != 1) {
    throw std::string("Output tensor must be 1D");
  } else if (out.dim[0] != in.dim[0]) {
    throw std::string("Size mismatch between input and output tensors");
  }

  hipLaunchConfig_t config = {};
  if(in.dim[1] < MAX_THREADS_PER_BLOCK) {
    config.blockDim.x = in.dim[1];
  } else {
    config.blockDim.x = MAX_THREADS_PER_BLOCK;
  }
  config.gridDim.x = in.dim[0];

  auto err = cudaLaunchKernelEx(&config, sum2DKernel<double>, out.mem, in.mem, in.dim[1]);
  if (err != hipSuccess) {
    throw std::string(hipGetErrorString(err));
  }
}