#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdint>
#include <libgpuc_cuda.hpp>
#include <string>

template <typename T>
class Variance {
public:
  T mean = 0;
  uint32_t n = 0;
  T m2 = 0;

  __device__ void comsume(T sample) {
    n++;
    T delta = sample - mean;
    mean += delta / n;
    m2 += delta * (sample - mean);
  }

  __device__ void merge(const Variance<T>& other) {
    if (other.n == 0) {
      return;
    }
    if (n == 0) {
      mean = other.mean;
      n = other.n;
      m2 = other.m2;
      return;
    }

    n = n + other.n;
    T delta = other.mean - mean;
    mean += delta * other.n / n;
    m2 += other.m2 + delta * delta * n * other.n / (n + other.n);
  }

  __device__ Variance<T> shfl_down(int offset) {
    Variance<T> other;
    other.mean = __shfl_down_sync(0xffffffff, mean, offset);
    other.n = __shfl_down_sync(0xffffffff, n, offset);
    other.m2 = __shfl_down_sync(0xffffffff, m2, offset);
    return other;
  }
};

template <typename T>
__global__ void variance2DKernel(T* out, T* in, uint32_t numCols) {
  uint32_t numThreads = blockDim.x;
  // uint32_t numRows = gridDim.y;
  uint32_t row = blockIdx.x;
  Variance<T> record{};
  for (uint32_t col = threadIdx.x; col < numCols; col += numThreads) {
    uint32_t idx = row * numCols + col;
    record.comsume(in[idx]);
  }
  __syncthreads();

  // Do warp reduction
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    record.merge(record.shfl_down(offset));
  }
  __syncthreads();

  uint8_t lane = threadIdx.x % warpSize;
  uint8_t warp = threadIdx.x / warpSize;

  __shared__ Variance<T> sdata[32];

  if (lane == 0) {
    sdata[warp] = record;
  }
  __syncthreads();

  if (warp == 0) {
    record = (lane < blockDim.x / warpSize) ? sdata[lane] : Variance<T>{};
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
      record.merge(record.shfl_down(offset));
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    out[row] = record.m2;
  }
}

void variance2DTensor(Tensor out, Tensor in) {
  if (in.ndim != 2) {
    throw std::string("Input tensor must be 2D");
  } else if (out.ndim != 1) {
    throw std::string("Output tensor must be 1D");
  } else if (out.dim[0] != in.dim[0]) {
    throw std::string("Size mismatch between input and output tensors");
  }

  hipLaunchConfig_t config = {};
  if(in.dim[1] < MAX_THREADS_PER_BLOCK) {
    config.blockDim.x = in.dim[1];
  } else {
    config.blockDim.x = MAX_THREADS_PER_BLOCK;
  }
  config.gridDim.x = in.dim[0];

  auto err = cudaLaunchKernelEx(&config, variance2DKernel<double>, out.mem, in.mem, in.dim[1]);
  if (err != hipSuccess) {
    throw std::string(hipGetErrorString(err));
  }
}