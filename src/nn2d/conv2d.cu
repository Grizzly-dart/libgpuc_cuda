#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdint>
#include <string>

#include "libgpuc_cuda.hpp"
#include "padding.hpp"

#define BLOCK_SIZE 16

// TODO if kernel size < 16, use shared memory
 
/// https://github.com/vdumoulin/conv_arithmetic/blob/master/README.md
template <typename T>
__global__ void conv2dKernel(T* output, T* input, T* kernel, uint32_t inChannels, uint32_t groups,
                             Dim2 inS, Dim2 kernS,
                             Dim2 padding, PaddingMode paddingMode, T pad, Dim2 stride, Dim2 dilation) {
  uint32_t kernNel = kernS.x * kernS.y;
  uint32_t outR = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t outC = blockIdx.x * blockDim.x + threadIdx.x;
  Dim2 outS = {x : gridDim.x * blockDim.x, y : gridDim.y * blockDim.y};
  uint32_t outId = blockIdx.z;
  uint32_t outChannels = gridDim.z;
  uint32_t groupLen = inChannels / groups;
  uint32_t firstInpChannelId = (outChannels / groups) / groupLen;

  if (outR < outS.y && outC < outS.x) {
    T value = 0;
    for (uint32_t kRow = 0; kRow < kernS.y; kRow++) {
      uint32_t inR = outR * stride.y + kRow * dilation.y;
      for (uint32_t kCol = 0; kCol < kernS.x; kCol++) {
        uint32_t inC = outC * stride.x + kCol * dilation.x;
        if (inR < inS.y + 2 * padding.y && inC < inS.x + 2 * padding.x) {
          for (uint32_t g = 0; g < groupLen; g++) {
            T* inputStart = input + (firstInpChannelId + g) * inS.x * inS.y;
            uint32_t kIdx = outId * groupLen + g;
            T inputValue = padder<T>(inputStart, inS, padding, paddingMode, pad, inC, inR);
            value += inputValue * kernel[kIdx * kernNel + kRow * kernS.y + kCol];
          }
        } else {
          assert(inR < inS.y + 2 * padding.y && inC < inS.x + 2 * padding.x);
        }
      }
    }
    output[outId * outS.x * outS.y + outR * outS.x + outC] = value;
  }
}

void conv2d(Tensor out, Tensor in, Tensor kernel, uint32_t groups,
            Dim2 padding, PaddingMode paddingMode, double pad, Dim2 stride, Dim2 dilation) {
  if (groups == 0) {
    groups = 1;
  }
  if (out.ndim != in.ndim) {
    throw std::string("out and in should have the same number of dimensions");
  } else if (getTensorB(out) != getTensorB(in)) {
    throw std::string("out and in should have the same batch size");
  }
  const uint32_t outChannels = getTensorC(out);
  const uint32_t inChannels = getTensorC(in);
  if (groups > 1) {
    if (outChannels % groups != 0) {
      throw std::string("out channels should be divisible by groups");
    }
    if (inChannels % groups != 0) {
      throw std::string("in channels should be divisible by groups");
    }
  }
  if (kernel.ndim != 4) {
    throw std::string("kernel should have 4 dimensions");
  }
  if (kernel.dim[0] != outChannels) {
    throw std::string("kernel should have the same number of channels as out");
  } else if (kernel.dim[1] != in.dim[1] / groups) {
    throw std::string("kernel should have the same number of channels as in");
  }
  // Compute output size based on padding, stride, dilation
  uint32_t outM = (getTensorM(in) + 2 * padding.y - dilation.y * (kernel.dim[2] - 1) - 1) / stride.y + 1;
  uint32_t outN = (getTensorN(in) + 2 * padding.x - dilation.x * (kernel.dim[3] - 1) - 1) / stride.x + 1;
  if (outM != getTensorM(out) || outN != getTensorN(out)) {
    throw std::string("output size is not correct");
  }

  for (uint32_t batch = 0; batch < getTensorB(out); batch++) {
    double* outPtr = out.mem + batch * getTensorM(out) * getTensorN(out) * getTensorC(out);
    double* inPtr = in.mem + batch * getTensorM(in) * getTensorN(in) * getTensorC(in);
    hipLaunchConfig_t config = {};
    if (outN < BLOCK_SIZE) {
      config.blockDim.x = outN;
      config.gridDim.x = 1;
    } else {
      config.blockDim.x = BLOCK_SIZE;
      config.gridDim.x = (outN + BLOCK_SIZE - 1) / BLOCK_SIZE;
    }
    if (outM < BLOCK_SIZE) {
      config.blockDim.y = outM;
      config.gridDim.y = 1;
    } else {
      config.blockDim.y = BLOCK_SIZE;
      config.gridDim.y = (outM + BLOCK_SIZE - 1) / BLOCK_SIZE;
    }
    config.blockDim.z = outChannels;
    auto err = cudaLaunchKernelEx(&config, conv2dKernel<double>, outPtr, inPtr, kernel.mem, inChannels, groups,
                                  Dim2{x : uint32_t(getTensorM(in)), y : uint32_t(getTensorN(in))}, Dim2{x : uint32_t(kernel.dim[2]), y : uint32_t(kernel.dim[3])},
                                  padding, paddingMode, pad, stride, dilation);
    if (err != hipSuccess) {
      throw std::string(hipGetErrorString(err));
    }
  }
}