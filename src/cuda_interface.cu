#include <hip/hip_runtime.h>
#include <memory.h>

#include <cstdint>
#include <libgpuc_cuda.hpp>
#include <string>

const char* libtcCudaCreateStream(libtcCudaStream& ret, int32_t device) {
  auto err = hipSetDevice(device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  hipStream_t stream;
  err = hipStreamCreate(&stream);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  ret.stream = stream;
  ret.device = device;
  return nullptr;
}

const char* libtcCudaDestroyStream(libtcCudaStream& ret) {
  auto err = hipSetDevice(ret.device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  err = hipStreamDestroy(static_cast<hipStream_t>(ret.stream));
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  return nullptr;
}

const char* libtcCudaAlloc(libtcCudaStream& stream, void** mem, uint64_t size) {
  auto err = hipSetDevice(stream.device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  err = hipMallocAsync(mem, size, stream.stream);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  return nullptr;
}

const char* libtcCudaFree(libtcCudaStream& stream, void* ptr) {
  auto err = hipSetDevice(stream.device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  err = hipFreeAsync(ptr, stream.stream);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  return nullptr;
}

const char* libtcCudaMemcpy(libtcCudaStream& stream, void* dst, void* src, uint64_t size) {
  auto err = hipSetDevice(stream.device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  err = hipMemcpyAsync(dst, src, size, hipMemcpyDefault, stream.stream);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  return nullptr;
}

const char* libtcCudaGetMemInfo(libtcCudaMemInfo& memInfo, int32_t device) {
  auto err = hipSetDevice(device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  err = hipMemGetInfo(&memInfo.free, &memInfo.total);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  return nullptr;
}

const char* libtcCudaGetDeviceProps(libtcDeviceProps& ret, int32_t device) {
  hipDeviceProp_t props;
  auto err = hipGetDeviceProperties(&props, device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  ret.totalGlobalMem = props.totalGlobalMem;
  ret.totalConstMem = props.totalConstMem;
  ret.sharedMemPerBlock = props.sharedMemPerBlock;
  ret.reservedSharedMemPerBlock = props.reservedSharedMemPerBlock;
  ret.sharedMemPerMultiprocessor = props.sharedMemPerMultiprocessor;
  ret.warpSize = static_cast<uint32_t>(props.warpSize);
  ret.multiProcessorCount = static_cast<uint32_t>(props.multiProcessorCount);
  ret.maxThreadsPerMultiProcessor = static_cast<uint32_t>(props.maxThreadsPerMultiProcessor);
  ret.maxThreadsPerBlock = static_cast<uint32_t>(props.maxThreadsPerBlock);
  ret.maxBlocksPerMultiProcessor = static_cast<uint32_t>(props.maxBlocksPerMultiProcessor);
  ret.l2CacheSize = static_cast<uint32_t>(props.l2CacheSize);
  ret.memPitch = static_cast<uint32_t>(props.memPitch);
  ret.memoryBusWidth = static_cast<uint32_t>(props.memoryBusWidth);
  ret.pciBusID = static_cast<uint32_t>(props.pciBusID);
  ret.pciDeviceID = static_cast<uint32_t>(props.pciDeviceID);
  ret.pciDomainID = static_cast<uint32_t>(props.pciDomainID);
  return nullptr;
}

void libtcFree(void* ptr) {
  free(ptr);
}

void* libtcRealloc(void* ptr, uint64_t size) {
  return realloc(ptr, size);
}

void libtcMemcpy(void* dst, void* src, uint64_t size) {
  memcpy(dst, src, size);
}