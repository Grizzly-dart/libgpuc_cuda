#include <hip/hip_runtime.h>
#include <memory.h>

#include <cstdint>
#include <libgpuc_cuda.hpp>
#include <string>

void* libtcCudaAlloc(uint64_t size, int32_t device) {
  auto err = hipSetDevice(device);
  if (err != hipSuccess) {
    throw std::string(hipGetErrorString(err));
  }
  void* ret;
  err = hipMalloc(&ret, size);
  if (err != hipSuccess) {
    printf("Error allocating: %s\n", hipGetErrorString(err));
    throw std::string(hipGetErrorString(err));
  }
  return ret;
}

void libtcCudaFree(void* ptr, int32_t device) {
  auto err = hipSetDevice(device);
  if (err != hipSuccess) {
    throw std::string(hipGetErrorString(err));
  }
  hipFree(ptr);
}

void libtcCudaMemcpy(void* dst, void* src, uint64_t size, uint8_t dir, int32_t device) {
  auto err = hipSetDevice(device);
  if (err != hipSuccess) {
    printf("Error:%d: %s\n", device, hipGetErrorString(err));
    fflush(stdout);
    throw std::string(hipGetErrorString(err));
  }
  printf("Copying %lu bytes from %p to %p %d\n", size, src, dst, dir);
  err = hipMemcpy(dst, src, size, hipMemcpyKind(dir));
  if (err != hipSuccess) {
    printf("Error: %s\n", hipGetErrorString(err));
    fflush(stdout);
    throw std::string(hipGetErrorString(err));
  }
  printf("Copied %lu bytes from %p to %p\n", size, src, dst);
}

libtcDeviceProps libtcCudaGetDeviceProps(int32_t device) {
  hipDeviceProp_t props;
  auto err = hipGetDeviceProperties(&props, device);
  if (err != hipSuccess) {
    printf("Error: %s\n", hipGetErrorString(err));
    fflush(stdout);
    throw err;
  }
  return libtcDeviceProps{
    totalGlobalMem : props.totalGlobalMem,
    totalConstMem : props.totalConstMem,
    sharedMemPerBlock : props.sharedMemPerBlock,
    reservedSharedMemPerBlock : props.reservedSharedMemPerBlock,
    sharedMemPerMultiprocessor : props.sharedMemPerMultiprocessor,
    warpSize : static_cast<uint32_t>(props.warpSize),
    multiProcessorCount : static_cast<uint32_t>(props.multiProcessorCount),
    maxThreadsPerMultiProcessor : static_cast<uint32_t>(props.maxThreadsPerMultiProcessor),
    maxThreadsPerBlock : static_cast<uint32_t>(props.maxThreadsPerBlock),
    maxBlocksPerMultiProcessor : static_cast<uint32_t>(props.maxBlocksPerMultiProcessor),
    l2CacheSize : static_cast<uint32_t>(props.l2CacheSize),
    memPitch : static_cast<uint32_t>(props.memPitch),
    memoryBusWidth : static_cast<uint32_t>(props.memoryBusWidth),
    pciBusID : static_cast<uint32_t>(props.pciBusID),
    pciDeviceID : static_cast<uint32_t>(props.pciDeviceID),
    pciDomainID : static_cast<uint32_t>(props.pciDomainID),
  };
}

void* libtcRealloc(void* ptr, uint64_t size) {
  return realloc(ptr, size);
}

void libtcMemcpy(void* dst, void* src, uint64_t size) {
  memcpy(dst, src, size);
}