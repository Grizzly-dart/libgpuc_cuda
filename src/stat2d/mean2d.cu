#include "hip/hip_runtime.h"
#include <string>
#include <cstdint>

#include <hip/hip_runtime.h>

#include <libgpuc_cuda.hpp>
#include <reducers.hpp>

template <typename T>
__global__ void mean2DKernel(T* out, T* in, uint32_t numCols) {
  uint32_t numThreads = blockDim.x;
  // uint32_t numRows = gridDim.y;
  uint32_t row = blockIdx.x;
  Mean<T> record{};
  for (uint32_t col = threadIdx.x; col < numCols; col += numThreads) {
    uint32_t idx = row * numCols + col;
    record.consume(in[idx]);
  }
  __syncthreads();

  // Do warp reduction
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    record.merge(record.shfl_down(offset));
  }
  __syncthreads();

  uint8_t lane = threadIdx.x % warpSize;
  uint8_t warp = threadIdx.x / warpSize;

  __shared__ Mean<T> sdata[32];

  if (lane == 0) {
    sdata[warp] = record;
  }
  __syncthreads();

  if (warp == 0) {
    record = (lane < blockDim.x / warpSize) ? sdata[lane] : Mean<T>{};
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
      record.merge(record.shfl_down(offset));
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    out[row] = record.mean;
  }
}

void mean2DTensor(Tensor out, Tensor in) {
  if (in.ndim != 2) {
    throw std::string("Input tensor must be 2D");
  } else if (out.ndim != 1) {
    throw std::string("Output tensor must be 1D");
  } else if (out.dim[0] != in.dim[0]) {
    throw std::string("Size mismatch between input and output tensors");
  }

  hipLaunchConfig_t config = {};
  if(in.dim[1] < MAX_THREADS_PER_BLOCK) {
    config.blockDim.x = in.dim[1];
  } else {
    config.blockDim.x = MAX_THREADS_PER_BLOCK;
  }
  config.gridDim.x = in.dim[0];

  auto err = cudaLaunchKernelEx(&config, mean2DKernel<double>, out.mem, in.mem, in.dim[1]);
  if (err != hipSuccess) {
    throw std::string(hipGetErrorString(err));
  }
}