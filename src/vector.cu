#include "hip/hip_runtime.h"
#include <string>
#include <cstdint>

#include <hip/hip_runtime.h>

#include <libgpuc_cuda.hpp>

// TODO implement stride and split
/// Adds two tensors
template<typename T>
__global__ void add2DKernel(T* out, const T* in1, const T* in2, uint32_t n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i >= n) return;
  out[i] = in1[i] + in2[i];
}

const char* libtcCudaAddCkern(libtcCudaStream& stream, double* out, const double* in1, const double* in2, uint32_t n) {
  uint32_t threads = n;
  uint32_t blocks = 1;
  if (n > MAX_THREADS_PER_BLOCK) {
    threads = MAX_THREADS_PER_BLOCK;
    blocks = (n + threads - 1) / threads;
  }
  hipLaunchConfig_t config = {
    .stream = stream.stream,
  };
  config.blockDim.x = threads;
  config.gridDim.x = blocks;
  auto err = hipSetDevice(stream.device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  err = cudaLaunchKernelEx(&config, add2DKernel<double>, out, in1, in2, n);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  // TODO remove
  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  return nullptr;
}

void add2D(Tensor out, Tensor in1, Tensor in2) {
  uint32_t n = getTensorNel(in1);
  if (n != getTensorNel(in2) || n != getTensorNel(out))
    throw std::string("Size mismatch");

  uint32_t threads = n;
  uint32_t blocks = 1;
  if (n > MAX_THREADS_PER_BLOCK) {
    threads = MAX_THREADS_PER_BLOCK;
    blocks = (n + threads - 1) / threads;
  }
  hipLaunchConfig_t config = {};
  config.blockDim.x = threads;
  config.gridDim.x = blocks;
  auto err = cudaLaunchKernelEx(&config, add2DKernel<double>, out.mem, in1.mem, in2.mem, n);
  if (err != hipSuccess) {
    throw std::string(hipGetErrorString(err));
  }
}