#include "hip/hip_runtime.h"
#include <string>
#include <cstdint>

#include <hip/hip_runtime.h>

#include <libgpuc_cuda.hpp>

/// Adds two vectors
template<typename T>
__global__ void add2DKernel(T* out, const T* in1, const T* in2, uint32_t n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i >= n) return;
  out[i] = in1[i] + in2[i];
}

void ewiseF64Add2(Tensor out, Tensor in1, Tensor in2) {
  uint32_t n = getTensorNel(in1);
  if (n != getTensorNel(in2) || n != getTensorNel(out))
    throw std::string("Size mismatch");

  uint32_t threads = n;
  uint32_t blocks = 1;
  if (n > MAX_THREADS_PER_BLOCK) {
    threads = MAX_THREADS_PER_BLOCK;
    blocks = (n + threads - 1) / threads;
  }
  hipLaunchConfig_t config = {};
  config.blockDim.x = threads;
  config.gridDim.x = blocks;
  auto err = cudaLaunchKernelEx(&config, add2DKernel<double>, out.mem, in1.mem, in2.mem, n);
  if (err != hipSuccess) {
    throw std::string(hipGetErrorString(err));
  }
}