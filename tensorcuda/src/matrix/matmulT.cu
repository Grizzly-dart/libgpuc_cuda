#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <cstdint>
#include <tensorcuda.hpp>
#include <string>

// TODO should this be warpSize?
#define TILE_SIZE 32

template <typename T>
__global__ void matmulT(
    T *out, T *inp1, T *inp2T, uint32_t m, uint32_t n, uint32_t k
) {
  int outRow = blockIdx.y * blockDim.y + threadIdx.y;
  int outCol = blockIdx.x * blockDim.x + threadIdx.x;
  int batches = blockIdx.z;

  inp1 += batches * m * n;
  inp2T += batches * n * k;
  out += batches * m * k;

  // TILE_SIZE+1 to avoid shared memory bank conflicts
  __shared__ T tile1[TILE_SIZE][TILE_SIZE + 1];
  __shared__ T tile2[TILE_SIZE][TILE_SIZE + 1];

  Dim2 inp2TileStart{
      blockIdx.x * blockDim.x + threadIdx.y, blockIdx.y * blockDim.y + threadIdx.x
  };

  T sum = 0.0;
  for (int i = 0; i < n; i += TILE_SIZE) {
    if (outRow < m && i + threadIdx.x < n) {
      T val = inp1[outRow * n + i + threadIdx.x];
      tile1[threadIdx.y][threadIdx.x] = val;
    }
    {
      uint32_t row = inp2TileStart.r;
      uint32_t col = inp2TileStart.c + i;
      if (row < k && col < n) {
        T val = inp2T[row * n + col];
        tile2[threadIdx.y][threadIdx.x] = val;
      }
    }
    __syncthreads();

    for (int j = 0; j < TILE_SIZE; ++j) {
      if (outRow < m && outCol < k && i + j < n) {
        sum += tile1[threadIdx.y][j] * tile2[threadIdx.x][j];
      }
    }
    __syncthreads();
  }

  if (outRow < m && outCol < k) {
    out[outRow * k + outCol] = sum;
  }
}

char const *tcuMatMulT(
    tcuStream &stream, double *out, double *inp1, double *inp2T,
    uint32_t m, uint32_t n, uint32_t k, uint32_t batches
) {
  auto err = hipSetDevice(stream.device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }

  hipLaunchConfig_t config = {
      .stream = stream.stream,
  };
  uint32_t max = m > k ? m : k;
  max = max > n ? max : n;
  if (max < TILE_SIZE) {
    config.blockDim = dim3(max, max);
  } else {
    config.blockDim = dim3(TILE_SIZE, TILE_SIZE);
  }
  config.gridDim.x = (k + config.blockDim.x - 1) / config.blockDim.x;
  config.gridDim.y = (m + config.blockDim.y - 1) / config.blockDim.y;
  config.gridDim.z = batches;
  err = cudaLaunchKernelEx(&config, matmulT<double>, out, inp1, inp2T, m, n, k);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  return nullptr;
}