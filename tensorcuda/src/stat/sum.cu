#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdint>
#include <libgpuc_cuda.hpp>
#include <string>

template <typename T, typename I>
__global__ void sum(T *out, I *inp, uint64_t nel) {
  uint32_t numThreads = blockDim.x * gridDim.x;
  uint32_t thId = threadIdx.x + blockIdx.x * blockDim.x;

  T sum = 0;
  for (uint64_t col = thId; col < nel; col += numThreads) {
    sum += inp[col];
  }
  __syncthreads();

  // Do warp reduction
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    sum += __shfl_down_sync(0xffffffff, sum, offset);
  }
  __syncthreads();

  uint8_t lane = threadIdx.x % warpSize;
  uint8_t warp = threadIdx.x / warpSize;

  __shared__ T sdata[32];
  if (warp == 0) {
    sdata[threadIdx.x] = 0;
  }
  __syncthreads();

  if (lane == 0) {
    sdata[warp] = sum;
  }
  __syncthreads();

  if (warp == 0) {
    sum = sdata[lane];
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
      sum += __shfl_down_sync(0xffffffff, sum, offset);
    }
  } else {
    sum = 0;
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    atomicAdd(out, sum);
  }
}