#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdint>
#include <libgpuc_cuda.hpp>
#include <reducers.hpp>
#include <string>

template <typename O, typename I>
__global__ void variance(
    O *out, I *inp, uint64_t nel, uint64_t correction
) {
  uint32_t numThreads = blockDim.x;

  inp += row * numCols;

  Variance<double> record{};
  for (uint64_t col = threadIdx.x; col < numCols; col += numThreads) {
    record.consume(inp[col]);
  }
  __syncthreads();

  // Do warp reduction
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    record.merge(record.shfl_down(offset));
  }
  __syncthreads();

  uint8_t lane = threadIdx.x % warpSize;
  uint8_t warp = threadIdx.x / warpSize;

  __shared__ Variance<double> sdata[32];
  if (warp == 0) {
    sdata[threadIdx.x] = {0};
  }
  __syncthreads();

  if (lane == 0) {
    sdata[warp] = record;
  }
  __syncthreads();

  if (warp == 0) {
    record = sdata[lane];
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
      record.merge(record.shfl_down(offset));
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    O val = record.m2 / (numCols - correction);
    *out = val;
  }
}