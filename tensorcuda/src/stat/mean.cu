#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdint>
#include <tensorcuda.hpp>
#include <reducers.hpp>
#include <string>

template <typename I> __global__ void mean(double *out, I *inp, uint64_t nel) {
  uint32_t numThreads = blockDim.x;

  Mean<double> record{};
  for (uint64_t col = threadIdx.x; col < nel; col += numThreads) {
    record.consume(inp[col]);
  }
  __syncthreads();

  // Do warp reduction
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    record.merge(record.shfl_down(offset));
  }
  __syncthreads();

  uint8_t lane = threadIdx.x % warpSize;
  uint8_t warp = threadIdx.x / warpSize;

  __shared__ Mean<double> sdata[32];
  if (warp == 0) {
    sdata[threadIdx.x] = {0};
  }
  __syncthreads();

  if (lane == 0) {
    sdata[warp] = record;
  }
  __syncthreads();

  if (warp == 0) {
    record = sdata[lane];
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
      record.merge(record.shfl_down(offset));
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    *out = record.mean;
  }
}

const char *tcuMean(
    tcuStream &stream, double *out, void *inp, uint64_t nel, dtype inpType
) {
  auto err = hipSetDevice(stream.device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  hipDeviceProp_t prop;
  err = hipGetDeviceProperties(&prop, stream.device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  hipLaunchConfig_t config = {
      .stream = stream.stream,
  };
  if (nel < prop.maxThreadsPerBlock) {
    config.blockDim.x = nel;
  } else {
    config.blockDim.x = prop.maxThreadsPerBlock;
  }

  if (inpType == dtype::f64) {
    err = cudaLaunchKernelEx(&config, mean<double>, out, (double *)inp, nel);
  } else if (inpType == dtype::f32) {
    err = cudaLaunchKernelEx(&config, mean<float>, out, (float *)inp, nel);
  } else if (inpType == dtype::i64) {
    err = cudaLaunchKernelEx(&config, mean<int64_t>, out, (int64_t *)inp, nel);
  } else if (inpType == dtype::i32) {
    err = cudaLaunchKernelEx(&config, mean<int32_t>, out, (int32_t *)inp, nel);
  } else if (inpType == dtype::i16) {
    err = cudaLaunchKernelEx(&config, mean<int16_t>, out, (int16_t *)inp, nel);
  } else if (inpType == dtype::i8) {
    err = cudaLaunchKernelEx(&config, mean<int8_t>, out, (int8_t *)inp, nel);
  } else if (inpType == dtype::u64) {
    err =
        cudaLaunchKernelEx(&config, mean<uint64_t>, out, (uint64_t *)inp, nel);
  } else if (inpType == dtype::u32) {
    err =
        cudaLaunchKernelEx(&config, mean<uint32_t>, out, (uint32_t *)inp, nel);
  } else if (inpType == dtype::u16) {
    err =
        cudaLaunchKernelEx(&config, mean<uint16_t>, out, (uint16_t *)inp, nel);
  } else if (inpType == dtype::u8) {
    err = cudaLaunchKernelEx(&config, mean<uint8_t>, out, (uint8_t *)inp, nel);
  } else {
    return "Unsupported dtype";
  }
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  return nullptr;
}
