#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdint>
#include <libgpuc_cuda.hpp>
#include <reducers.hpp>
#include <string>

template <typename O, typename I>
__global__ void mean(O *out, I *inp, uint64_t nel) {
  uint32_t numThreads = blockDim.x;

  Mean<double> record{};
  for (uint64_t col = threadIdx.x; col < nel; col += numThreads) {
    record.consume(inp[col]);
  }
  __syncthreads();

  // Do warp reduction
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    record.merge(record.shfl_down(offset));
  }
  __syncthreads();

  uint8_t lane = threadIdx.x % warpSize;
  uint8_t warp = threadIdx.x / warpSize;

  __shared__ Mean<double> sdata[32];
  if (warp == 0) {
    sdata[threadIdx.x] = {0};
  }
  __syncthreads();

  if (lane == 0) {
    sdata[warp] = record;
  }
  __syncthreads();

  if (warp == 0) {
    record = sdata[lane];
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
      record.merge(record.shfl_down(offset));
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    *out = record.mean;
  }
}