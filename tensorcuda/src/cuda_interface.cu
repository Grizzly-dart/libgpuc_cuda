#include <hip/hip_runtime.h>
#include <memory.h>
#include <pthread.h> 

#include <cstdint>
#include <libgpuc_cuda.hpp>
#include <string>

const char* tcuCreateStream(tcuStream& ret, int32_t device) {
  auto err = hipSetDevice(device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  hipStream_t stream;
  err = hipStreamCreate(&stream);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  ret.stream = stream;
  ret.device = device;
  return nullptr;
}

const char* tcuDestroyStream(tcuStream& ret) {
  auto err = hipSetDevice(ret.device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  err = hipStreamDestroy(static_cast<hipStream_t>(ret.stream));
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  return nullptr;
}

typedef struct {
  tcuStream* stream;
  void (*callback)(const char*);
} syncStreamArgs;

void syncStream(syncStreamArgs* args) {
  auto stream = args->stream;
  auto callback = args->callback;
  free(args);
  auto err = hipSetDevice(stream->device);
  if (err != hipSuccess) {
    callback(hipGetErrorString(err));
    return;
  }
  err = hipStreamSynchronize(static_cast<hipStream_t>(stream->stream));
  if (err != hipSuccess) {
    callback(hipGetErrorString(err));
    return;
  }
  callback(nullptr);
  pthread_exit(NULL);
}

const char* tcuSyncStream(tcuStream* stream, void (*callback)(const char*)) {
  auto args = (syncStreamArgs*)(malloc(sizeof(syncStreamArgs)));
  args->stream = stream;
  args->callback = callback;

  pthread_attr_t attr;
  int rc = pthread_attr_init(&attr);                                               
  if (rc == -1) {                                                              
    return "cudaStreamSync: error in pthread_attr_init";                                                                  
  }                                                                 
  rc = pthread_attr_setdetachstate(&attr, 1);                                
  if (rc == -1) {
    return "cudaStreamSync: error in pthread_attr_setdetachstate";                                                               
  } 

  pthread_t thread;
  pthread_create(&thread, NULL, (void *(*)(void*))syncStream, args);
  return nullptr;
}

const char* tcuAlloc(tcuStream& stream, void** mem, uint64_t size) {
  auto err = hipSetDevice(stream.device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  err = hipMallocAsync(mem, size, stream.stream);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  return nullptr;
}

const char* tcuFree(tcuStream& stream, void* ptr) {
  auto err = hipSetDevice(stream.device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  err = hipFreeAsync(ptr, stream.stream);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  return nullptr;
}

const char* tcuMemcpy(tcuStream& stream, void* dst, void* src, uint64_t size) {
  auto err = hipSetDevice(stream.device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  err = hipMemcpyAsync(dst, src, size, hipMemcpyDefault, stream.stream);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  return nullptr;
}

const char* tcuGetMemInfo(tcuMemInfo& memInfo, int32_t device) {
  auto err = hipSetDevice(device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  err = hipMemGetInfo(&memInfo.free, &memInfo.total);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  return nullptr;
}

const char* tcuGetDeviceProps(tcuDeviceProps& ret, int32_t device) {
  hipDeviceProp_t props;
  auto err = hipGetDeviceProperties(&props, device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  ret.totalGlobalMem = props.totalGlobalMem;
  ret.totalConstMem = props.totalConstMem;
  ret.sharedMemPerBlock = props.sharedMemPerBlock;
  ret.reservedSharedMemPerBlock = props.reservedSharedMemPerBlock;
  ret.sharedMemPerMultiprocessor = props.sharedMemPerMultiprocessor;
  ret.warpSize = static_cast<uint32_t>(props.warpSize);
  ret.multiProcessorCount = static_cast<uint32_t>(props.multiProcessorCount);
  ret.maxThreadsPerMultiProcessor = static_cast<uint32_t>(props.maxThreadsPerMultiProcessor);
  ret.maxThreadsPerBlock = static_cast<uint32_t>(props.maxThreadsPerBlock);
  ret.maxBlocksPerMultiProcessor = static_cast<uint32_t>(props.maxBlocksPerMultiProcessor);
  ret.l2CacheSize = static_cast<uint32_t>(props.l2CacheSize);
  ret.memPitch = static_cast<uint32_t>(props.memPitch);
  ret.memoryBusWidth = static_cast<uint32_t>(props.memoryBusWidth);
  ret.pciBusID = static_cast<uint32_t>(props.pciBusID);
  ret.pciDeviceID = static_cast<uint32_t>(props.pciDeviceID);
  ret.pciDomainID = static_cast<uint32_t>(props.pciDomainID);
  return nullptr;
}