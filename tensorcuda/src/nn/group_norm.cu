#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdint>
#include <libgpuc_cuda.hpp>
#include <string>

template <typename T>
__global__ void rowNorm(
    T *input, T *output, uint64_t elements, 
    double epsilon
) {
  // TODO
}

const char *libtcRowNorm(
    libtcCudaStream &stream, void *out, void *inp, uint64_t cols, uint32_t rows,
    double epsilon
) {
  auto err = hipSetDevice(stream.device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  hipDeviceProp_t props;
  err = hipGetDeviceProperties(&props, stream.device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }

  hipLaunchConfig_t config = {
      .stream = stream.stream,
  };
  // TODO
}
