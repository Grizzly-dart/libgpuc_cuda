#include "hip/hip_runtime.h"
#include <string>
#include <cstdint>

#include <hip/hip_runtime.h>

#include <libgpuc_cuda.hpp>

// TODO implement stride and split
/// Adds two tensors
template<typename T>
__global__ void add2Kernel(T* out, const T* in1, const T* in2, uint64_t n) {
  uint32_t numThreads = blockDim.x * gridDim.x;
  uint32_t thId = threadIdx.x + blockIdx.x * blockDim.x;

  for(uint64_t i = thId; i < n; i += numThreads) {
    out[i] = in1[i] + in2[i];
  }
}

const char* libtcCudaAdd2(libtcCudaStream& stream, double* out, const double* in1, const double* in2, uint64_t n) {
  auto err = hipSetDevice(stream.device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  hipDeviceProp_t props;
  err = hipGetDeviceProperties(&props, stream.device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  uint32_t numThreads = props.multiProcessorCount * 128;
  if(numThreads > n) {
    numThreads = n;
  }
  
  hipLaunchConfig_t config = {
    .stream = stream.stream,
  };
  if(numThreads < props.maxThreadsPerBlock) {
    config.blockDim.x = numThreads;
    config.gridDim.x = 1;
  } else {
    config.blockDim.x = props.maxThreadsPerBlock;
    config.gridDim.x = (n + props.maxThreadsPerBlock - 1) / props.maxThreadsPerBlock;
  }

  err = cudaLaunchKernelEx(&config, add2Kernel<double>, out, in1, in2, n);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  // TODO remove
  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  return nullptr;
}