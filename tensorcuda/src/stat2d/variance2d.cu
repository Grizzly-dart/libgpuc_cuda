#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdint>
#include <libgpuc_cuda.hpp>
#include <reducers.hpp>
#include <string>

template <typename O, typename I>
__global__ void variance2d(O* out, I* inp, uint64_t numCols, uint64_t correction, bool calcStd) {
  uint32_t numThreads = blockDim.x;
  uint32_t row = blockIdx.x;
  
  inp += row * numCols;

  Variance<double> record{};
  for (uint64_t col = threadIdx.x; col < numCols; col += numThreads) {
    if (col < numCols) {
      record.consume(inp[col]);
    }
  }
  __syncthreads();

  // Do warp reduction
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    record.merge(record.shfl_down(offset));
  }
  __syncthreads();

  uint8_t lane = threadIdx.x % warpSize;
  uint8_t warp = threadIdx.x / warpSize;

  __shared__ Variance<double> sdata[32];
  if (warp == 0) {
    sdata[threadIdx.x] = {0};
  }
  __syncthreads();

  if (lane == 0) {
    sdata[warp] = record;
  }
  __syncthreads();

  if (warp == 0) {
    record = sdata[lane];
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
      record.merge(record.shfl_down(offset));
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    O val = record.m2 / (numCols - correction);
    if(calcStd) {
      val = sqrt(val);
    }
    out[row] = val;
  }
}

const char *libtcVariance2d(
    libtcCudaStream &stream, void *out, void *inp, uint32_t rows, uint64_t cols, uint64_t correction, bool calcStd, dtype outType, dtype inpType
) {
  auto err = hipSetDevice(stream.device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  hipDeviceProp_t props;
  err = hipGetDeviceProperties(&props, stream.device);
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }

  hipLaunchConfig_t config = {
      .stream = stream.stream,
  };
  if (cols < 1024) {
    config.blockDim.x = cols;
  } else {
    config.blockDim.x = 1024;
  }
  config.gridDim.x = rows;

  if(outType == dtype::f64) {
    if(inpType == dtype::f64) {
      err = cudaLaunchKernelEx(&config, variance2d<double, double>, (double*)out, (double*)inp, cols, correction, calcStd);
    } else if(inpType == dtype::f32) {
      err = cudaLaunchKernelEx(&config, variance2d<double, float>, (double*)out, (float*)inp, cols, correction, calcStd);
    } else if(inpType == dtype::i64) {
      err = cudaLaunchKernelEx(&config, variance2d<double, int64_t>, (double*)out, (int64_t*)inp, cols, correction, calcStd);
    } else if(inpType == dtype::i32) {
      err = cudaLaunchKernelEx(&config, variance2d<double, int32_t>, (double*)out, (int32_t*)inp, cols, correction, calcStd);
    } else if(inpType == dtype::i16) {
      err = cudaLaunchKernelEx(&config, variance2d<double, int16_t>, (double*)out, (int16_t*)inp, cols, correction, calcStd);
    } else if(inpType == dtype::i8) {
      err = cudaLaunchKernelEx(&config, variance2d<double, int8_t>, (double*)out, (int8_t*)inp, cols, correction, calcStd);
    } else if(inpType == dtype::u64) {
      err = cudaLaunchKernelEx(&config, variance2d<double, uint64_t>, (double*)out, (uint64_t*)inp, cols, correction, calcStd);
    } else if(inpType == dtype::u32) {
      err = cudaLaunchKernelEx(&config, variance2d<double, uint32_t>, (double*)out, (uint32_t*)inp, cols, correction, calcStd);
    } else if(inpType == dtype::u16) {
      err = cudaLaunchKernelEx(&config, variance2d<double, uint16_t>, (double*)out, (uint16_t*)inp, cols, correction, calcStd);
    } else if(inpType == dtype::u8) {
      err = cudaLaunchKernelEx(&config, variance2d<double, uint8_t>, (double*)out, (uint8_t*)inp, cols, correction, calcStd);
    } else {
      return "Unsupported input type";
    }
  } else if(outType == dtype::f32) {
    if(inpType == dtype::f64) {
      err = cudaLaunchKernelEx(&config, variance2d<float, double>, (float*)out, (double*)inp, cols, correction, calcStd);
    } else if(inpType == dtype::f32) {
      err = cudaLaunchKernelEx(&config, variance2d<float, float>, (float*)out, (float*)inp, cols, correction, calcStd);
    } else if(inpType == dtype::i64) {
      err = cudaLaunchKernelEx(&config, variance2d<float, int64_t>, (float*)out, (int64_t*)inp, cols, correction, calcStd);
    } else if(inpType == dtype::i32) {
      err = cudaLaunchKernelEx(&config, variance2d<float, int32_t>, (float*)out, (int32_t*)inp, cols, correction, calcStd);
    } else if(inpType == dtype::i16) {
      err = cudaLaunchKernelEx(&config, variance2d<float, int16_t>, (float*)out, (int16_t*)inp, cols, correction, calcStd);
    } else if(inpType == dtype::i8) {
      err = cudaLaunchKernelEx(&config, variance2d<float, int8_t>, (float*)out, (int8_t*)inp, cols, correction, calcStd);
    } else if(inpType == dtype::u64) {
      err = cudaLaunchKernelEx(&config, variance2d<float, uint64_t>, (float*)out, (uint64_t*)inp, cols, correction, calcStd);
    } else if(inpType == dtype::u32) {
      err = cudaLaunchKernelEx(&config, variance2d<float, uint32_t>, (float*)out, (uint32_t*)inp, cols, correction, calcStd);
    } else if(inpType == dtype::u16) {
      err = cudaLaunchKernelEx(&config, variance2d<float, uint16_t>, (float*)out, (uint16_t*)inp, cols, correction, calcStd);
    } else if(inpType == dtype::u8) {
      err = cudaLaunchKernelEx(&config, variance2d<float, uint8_t>, (float*)out, (uint8_t*)inp, cols, correction, calcStd);
    } else {
      return "Unsupported input type";
    }
  } else if(outType == dtype::f16) {
    return "Unsupported output type";
  } else {
    return "Unsupported output type";
  }
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  return nullptr;
}